#include <iostream>
#include <string.h>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <vector>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <ctime>
using namespace std;
//qsub -I -q coc-ice -l nodes=1:ppn=8:gpus=1,walltime=04:30:00,pmem=2gb
//qsub -I -q coc-ice -l nodes=1,walltime=02:30:00,pmem=2gb
const float PI = 3.14159265358979f;
//class for Complex number -------------------------------------------------------
class Complex {
		public:
			__device__ __host__ Complex() : real(0.0f), imag(0.0f){
			}

			__device__ __host__ Complex(float r, float i) : real(r), imag(i){

			}

			__device__ __host__ Complex(float r) : real(r), imag(0.0f){

			}

			__device__ __host__ Complex operator+ (const Complex& b) const{
				return Complex(real + b.real, imag + b.imag);
			}

			__device__ __host__ Complex operator- (const Complex& b) const{
				return Complex(real - b.real, imag - b.imag);
			}

			__device__ __host__ Complex operator* (const Complex& b) const{
				return Complex(real * b.real - imag * b.imag, real * b.imag + imag * b.real);
			}

			__device__ __host__ Complex Mag() const{
				return Complex(sqrt(real * real + imag * imag));
			}

			__device__ __host__ Complex Angle() const{
				return Complex(atan2(imag, real) * 360 / (2 * PI));
			}

			__device__ __host__ Complex Conj() const{
				return Complex(real, -imag);
			}

			void Print() const{
				if(imag == 0){
					cout << real;
				}else{
					cout << '(' << real << ',' << imag << ')' << endl;
				}
			}

		
			float real;
			float imag;
};

ostream& operator<< (ostream& os, const Complex& rhs) {
    Complex c(rhs);
    if(fabsf(rhs.imag) < 1e-10) c.imag = 0.0f;
    if(fabsf(rhs.real) < 1e-10) c.real = 0.0f;

    if(c.imag == 0) {
        os << c.real;
    }
    else {
        os << "(" << c.real << "," << c.imag << ")";
    }
    return os;
}

//class for input and output image---------------------------------------------------
class InputImage {
public:

    InputImage(const char* filename);
    int get_width() const;
    int get_height() const;

    //returns a pointer to the image data.  Note the return is a 1D
    //array which represents a 2D image.  The data for row 1 is
    //immediately following the data for row 0 in the 1D array
    Complex* get_image_data() const;

    //use this to save output from forward DFT
    void save_image_data(const char* filename, Complex* d, int w, int h);
    //use this to save output from reverse DFT
    void save_image_data_real(const char* filename, Complex* d, int w, int h);
    //use this to check mag
    void save_image_data_mag(const char* filename, Complex* d, int w, int h);

private:
    int w;
    int h;
    Complex* data;
};

InputImage::InputImage(const char* filename) {
    std::ifstream ifs(filename);
    if(!ifs) {
        std::cout << "Can't open image file " << filename << std::endl;
        exit(1);
    }

    ifs >> w >> h;
    data = new Complex[w * h];
    for(int r = 0; r < h; ++r) {
        for(int c = 0; c < w; ++c) {
            float real;
            ifs >> real;
            data[r * w + c] = Complex(real);
        }
    }
}

int InputImage::get_width() const {
    return w;
}

int InputImage::get_height() const {
    return h;
}

Complex* InputImage::get_image_data() const {
    return data;
}

void InputImage::save_image_data(const char *filename, Complex *d, int w, int h) {
    std::ofstream ofs(filename);
    if(!ofs) {
        std::cout << "Can't create output image " << filename << std::endl;
        return;
    }

    ofs << w << " " << h << std::endl;

    for(int r = 0; r < h; ++r) {
        for(int c = 0; c < w; ++c) {
            ofs << d[r * w + c] << " ";
        }
        ofs << std::endl;
    }
}

void InputImage::save_image_data_real(const char* filename, Complex* d, int w, int h) {
    std::ofstream ofs(filename);
    if(!ofs) {
        std::cout << "Can't create output image " << filename << std::endl;
        return;
    }

    ofs << w << " " << h << std::endl;

    for (int r = 0; r < h; ++r) {
        for (int c = 0; c < w; ++c) {
            ofs << d[r * w + c].real << " ";
        }
        ofs << std::endl;
    }
}

void InputImage::save_image_data_mag(const char* newFileName, Complex* d,int w, int h){


  std::ofstream ofs(newFileName);
  if (!ofs)
    {
      std::cout << "Can't create output image " << newFileName << std::endl;
      return;
    }
  ofs << w << " " << h << endl;
  for (int r = 0; r < h; ++r)
    { // for each row
      for (int c = 0; c < w; ++c)
        { // for each column
          ofs << d[r * w + c].Mag() << " ";
        }
      ofs << std::endl;
    }
}


//test,test


__global__ void reorder(Complex* a, Complex* b, int N){
    long idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N * N){
        int x = idx / N;
        int y = idx % N;
        unsigned r = 0; // reversed index;
        unsigned n = N;
        unsigned num = y;

        for(--n; n > 0; n >>= 1){
            r <<= 1;
            r |= (num & 0x1);
            num >>= 1;
        }

        y = r;

        b[x * N + y] = a[idx];
    }
    
}

__global__ void CountW(Complex* W, int N){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N/2){
        W[idx] = Complex(cos(2 * PI * idx / N), -sin(2 * PI * idx / N));
        W[idx + N/2] = Complex(-1) * W[idx];
    }
    //__syncthreads();
}

__global__ void CountWInverse(Complex* W, int N){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N/2){
        W[idx] = Complex(cos(2 * PI * idx / N), sin(2 * PI * idx / N));
        W[idx + N/2] = Complex(-1) * W[idx];
    }
    //__syncthreads();
}

__global__ void Tmatrix(Complex* a, int width, int height){
    long idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < width * height){
        int x = idx / width;
        int y = idx % width;
        if(y > x){
            Complex tmp = a[x * width + y];
            a[x * width + y] = a[y * width + x];
            a[y * width + x] = tmp;
        }
    }
     //__syncthreads();
}

// __global__ void TransformAnArrray(Complex* a, Complex*b, int N, Complex* W){//b is all 0 initially
//     long idx = threadIdx.x + blockIdx.x * blockDim.x;
//     int x = idx / N;// x th row//temp[]->b; H[] -> a[x * width + 0] ~ a[x * width + width - 1]
//     int y = idx % N;// y th element in x th row

//     if(idx < N * N ){//

//         int groupLen = 2;
//         int groupNum = N / 2;
        
//         while(groupLen <= N){//a or b should include x* width first or just use idx as index
//             int i = y / groupLen;
//             int j = y % groupLen;
//             b[idx] = a[x * N + i * groupLen + j % (groupLen/2)] + a[x * N + i * groupLen + j%(groupLen/2) + groupLen/2] * W[N*j/groupLen];
//             __syncthreads();
//             a[idx] = b[idx];// should be ok?
//             groupLen *= 2;
//             groupNum /= 2;
//             __syncthreads();
//         }

//     }//
// }

 __global__ void TransformAnArrray(Complex* a, Complex*b, int N, Complex* W, int groupLen, int groupNum){
    long idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N * N){
        int x = idx / N;// x th row//temp[]->b; H[] -> a[x * width + 0] ~ a[x * width + width - 1]
        int y = idx % N;// y th element in x th row

        int i = y / groupLen;
        int j = y % groupLen;

        b[idx] = a[x * N + i * groupLen + j % (groupLen/2)] + a[x * N + i * groupLen + j%(groupLen/2) + groupLen/2] * W[N*j/groupLen];

    }
    
 }

__global__ void ConvertAB(Complex* a, Complex*b, int N){
     long idx = threadIdx.x + blockIdx.x * blockDim.x;
     if(idx < N * N){
        a[idx] = b[idx];
     }

}

__global__ void CmpleteInverseT(Complex*a, int N){
    long idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N * N){
        a[idx] = Complex(1.0 / N) * a[idx];
        if(a[idx].Mag().real < 1e-10){
            a[idx] = Complex(0);
        }
    }
}




int main(int argc, const char * argv[]) {

        clock_t startTime, endTime;
        const char* type = argv[1];
		const char* filename = argv[2];
        const char* outputfile = argv[3];

        startTime = clock();
        
		InputImage Tower(filename);

		int Tower_height = Tower.get_height();
		int Tower_width = Tower.get_width();
        Complex* TowerData = Tower.get_image_data();// data itself// one diamention array
//test-------------------------------------
		// cout << "height = " << Tower_height << "\n" << "width = " << Tower_width << endl;
        // for(long i = 0; i < Tower_width * Tower_height; i++){
        //     printf("thread:%ld, content:%f\n", i, TowerData[i].real);
        // }
//test-------------------------------------

        Complex* d_a;
        Complex* d_b;
        Complex* d_w;

        chrono::steady_clock::time_point tStart;
        tStart = chrono::steady_clock::now();

        hipMalloc(&d_a, sizeof(Complex) * Tower_width * Tower_height);
        hipMalloc(&d_b, sizeof(Complex) * Tower_width * Tower_height);
        hipMalloc(&d_w, sizeof(Complex) * Tower_width);
        
        hipMemcpy(d_a, TowerData, sizeof(Complex)* Tower_width * Tower_height, hipMemcpyHostToDevice);

        reorder<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width);
        ConvertAB<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width);

        CountW<<<(Tower_width + 511) / 512, 512>>>(d_w, Tower_width);

        int groupLen = 2;
        int groupNum = Tower_width / 2;

        while(groupLen <= Tower_width){

            TransformAnArrray<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width, d_w, groupLen, groupNum);
            ConvertAB<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width);
            groupLen *= 2;
            groupNum /= 2;

        }
//1D test
        // cudaMemcpy(TowerData, d_a, sizeof(Complex) * Tower_width * Tower_height, cudaMemcpyDeviceToHost);
        // Tower.save_image_data_mag(outputfile, TowerData, Tower_width, Tower_height);
//1D test correct
//---------------------------------------------------------------------------------------------------------
        Tmatrix<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, Tower_width, Tower_height);
        reorder<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width);
        ConvertAB<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width);
//---------------------------------------------------------------------------------------------------------
        groupLen = 2;
        groupNum = Tower_width / 2;

        while(groupLen <= Tower_width){

            TransformAnArrray<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width, d_w, groupLen, groupNum);
            ConvertAB<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width);
            groupLen *= 2;
            groupNum /= 2;

        }

        

        Tmatrix<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, Tower_width, Tower_height);

        if(type[0] == 'f'){
            hipMemcpy(TowerData, d_a, sizeof(Complex) * Tower_width * Tower_height, hipMemcpyDeviceToHost);
            chrono::steady_clock::time_point tEnd = chrono::steady_clock::now();
            chrono::duration<double> time_span = chrono::duration_cast<chrono::duration<double> >(tEnd - tStart);
            cout << "Time ellipsed: " << time_span.count() << " seconds... \n";
            Tower.save_image_data(outputfile, TowerData, Tower_width, Tower_height);
        }else{
            reorder<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width);
            ConvertAB<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width);

            CountWInverse<<<(Tower_width + 511) / 512, 512>>>(d_w, Tower_width);// different W

            groupLen = 2;
            groupNum = Tower_width / 2;

            while(groupLen <= Tower_width){

                TransformAnArrray<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width, d_w, groupLen, groupNum);
                ConvertAB<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width);
                groupLen *= 2;
                groupNum /= 2;

            }

            CmpleteInverseT<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, Tower_width);

            Tmatrix<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, Tower_width, Tower_height);
            reorder<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width);
            ConvertAB<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width);

            groupLen = 2;
            groupNum = Tower_width / 2;

            while(groupLen <= Tower_width){

                TransformAnArrray<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width, d_w, groupLen, groupNum);
                ConvertAB<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, d_b, Tower_width);
                groupLen *= 2;
                groupNum /= 2;

            }

            CmpleteInverseT<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, Tower_width);

            Tmatrix<<<(Tower_width * Tower_height + 511) / 512, 512>>>(d_a, Tower_width, Tower_height);
            hipMemcpy(TowerData, d_a, sizeof(Complex) * Tower_width * Tower_height, hipMemcpyDeviceToHost);
            chrono::steady_clock::time_point tEnd = chrono::steady_clock::now();
            chrono::duration<double> time_span = chrono::duration_cast<chrono::duration<double> >(tEnd - tStart);
            cout << "Time ellipsed: " << time_span.count() << " seconds... \n";
            Tower.save_image_data(outputfile, TowerData, Tower_width, Tower_height);
        }

        

        //z.Print();

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_w);
        delete[] TowerData;
        endTime = clock();
        cout<<"Total time = "<<(double)(endTime - startTime)/CLOCKS_PER_SEC<<"s"<<endl;
        //delete TowerData?
        return 0;
}
